#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>

#include "cudaFluid.cuh"
#include "../kernel.h"

#ifdef BUILD_CUDA

__global__ void simulate_update_position_predict_position(
  int n,
  Vector3R *particle_positions, 
  Vector3R *particle_preditced_position, 
  Vector3R *particle_velocities, 
  const Vector3R external_accelerations, 
  REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    auto &positions_i = particle_positions[i];
    auto &velocities_i = particle_velocities[i];
    auto &preditced_positions_i = particle_preditced_position[i];
    if (i == 0) {
      printf("Thread 0 position: (%lf, %lf, %lf), acc: (%lf, %lf, %lf), delta_t: %lf\n", 
        positions_i.x, positions_i.y, positions_i.z,
        external_accelerations.x, external_accelerations.y, external_accelerations.z,
        delta_t
      );
    }
    velocities_i += external_accelerations * delta_t;
    preditced_positions_i = positions_i + velocities_i * delta_t;
  }
}

__global__ void calculate_lambda(
  int n,
  Vector3R *particle_positions,
  int *neighbor_search_results,
  int *neighbor_search_results_size_prefix_sum,
  REAL particle_mass,
  REAL density,
  REAL epsilon,
  REAL h,
  REAL *lambda
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    // line 10: calculate lambda
    const int neighbors_size_last = i == 0 ? 0 : neighbor_search_results_size_prefix_sum[i-1];
    const int neighbors_size_i = neighbor_search_results_size_prefix_sum[i]-neighbors_size_last;
    const int *neighbors_i = &neighbor_search_results[neighbors_size_last];
    // Eq 2
    REAL rho_i = 0;
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      rho_i += W_poly6(p_i-p_j, h);
    }
    // add itself
    rho_i += W_poly6(p_i-p_i, h);
    rho_i *= particle_mass;
    // Eq 1
    const REAL C_i = rho_i / density - 1.;
    REAL C_i_p_k_2_sum = 0;
    // Eq 8
    // if k = j
    REAL C_i_p_k_j_2 = 0;
    // if k = i
    Vector3R C_i_p_k_i;
    for (int jj = 0; jj < neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      const auto W_spiky_gradient_i_j = W_spiky_gradient(p_i-p_j, h) * (p_i-p_j);
      C_i_p_k_i += W_spiky_gradient_i_j;
      C_i_p_k_j_2 += W_spiky_gradient_i_j.norm2();
    }
    C_i_p_k_2_sum += C_i_p_k_i.norm2();
    C_i_p_k_2_sum /= pow(density, 2);
    lambda[i] = - C_i / (C_i_p_k_2_sum+epsilon);
  }
}

__global__ void calculate_delta_pi_and_collision_response(
  int num_particles,
  Vector3R *particle_positions,
  int *neighbor_search_results,
  int *neighbor_search_results_size_prefix_sum,
  Vector3R *delta_p,
  REAL n,
  REAL k,
  REAL h,
  REAL density,
  REAL *lambda,
  int nObjs,
  Plane_cuda *collision_objects
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    // line 13: calculate delta p_i
    const int neighbors_size_last = i == 0 ? 0 : neighbor_search_results_size_prefix_sum[i-1];
    const int neighbors_size_i = neighbor_search_results_size_prefix_sum[i]-neighbors_size_last;
    const int *neighbors_i = &neighbor_search_results[neighbors_size_last];
    delta_p[i] = 0;
    const auto lambda_i = lambda[i];
    // Eq 12
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      // Eq 13
      double s_corr = -k*pow(W_poly6(p_i-p_j, h)/W_poly6(0.3*h, h), n);
      delta_p[i] += (lambda_i+lambda[j]+s_corr) * W_spiky_gradient(p_i-p_j, h) 
        * (p_i-p_j);
    }
    delta_p[i] /= density;
    // line 14: collision detection and response
    // TODO: apply them
    for (int j = 0; j < nObjs; j++) {
      collision_objects[j].collide(particle_positions[i],delta_p[i]);
    }
  }
}

__global__ void update_predicted_positions(
  int num_particles,
  Vector3R *particle_predicted_positions,
  Vector3R *delta_p
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    particle_predicted_positions[i] += delta_p[i];
  }
}

__global__ void update_velocities(
  int num_particles,
  Vector3R *particle_positions,
  Vector3R *predicted_positions,
  Vector3R *particle_velocities,
  REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    particle_velocities[i] = (predicted_positions[i] - particle_positions[i]) / delta_t;
  }
}

__global__ void apply_XSPH_viscosity(
  int num_particles,
  Vector3R *particle_positions,
  Vector3R *particle_velocities,
  int *neighbor_search_results,
  int *neighbor_search_results_size_prefix_sum,
  REAL particle_mass, REAL h, REAL c, REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    const int neighbors_size_last = i == 0 ? 0 : neighbor_search_results_size_prefix_sum[i-1];
    const int neighbors_size_i = neighbor_search_results_size_prefix_sum[i]-neighbors_size_last;
    const int *neighbors_i = &neighbor_search_results[neighbors_size_last];
    // line 22: vorticity confinement and XSPH viscosity
    Vector3R f_vorticity;
    Vector3R omega_i;
    // Eq 17:
    Vector3R V_xsph;
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      const auto &p_ij = p_i-p_j;
      const auto &v_ij = particle_velocities[j] - particle_velocities[i];
      // the smallest |p_ij| with h=0.1 gives >100 so c has to correct it to ~1
      V_xsph  += v_ij * W_poly6(p_ij, h);
      omega_i += cross(v_ij, W_spiky_gradient(p_ij, h)*p_ij);
    }
    // TODO: vorticity
    // const auto &eta = ;
    // const auto &N = eta.unit();
    // f_vorticity = epsilon*cross(N, omega_i);
    V_xsph *= c;
    particle_velocities[i] += V_xsph + f_vorticity / particle_mass * delta_t;
  }
}

void copy_predicted_positions_to_position(
  REAL3 *particle_position,
  REAL3 *particle_preditced_position, 
  size_t N
) {
  hipMemcpy(particle_position, particle_preditced_position, N, hipMemcpyDeviceToDevice);
}

Fluid_cuda::Fluid_cuda(
  unique_ptr<vector<REAL3>> &&particle_positions,
  REAL h
): nsearch(h) {
  if (particle_positions == nullptr) {
    throw std::runtime_error("particle_positions == nullptr!");
  }
  this->particle_positions = std::move(particle_positions);
}

Fluid_cuda::~Fluid_cuda(){
  hipFree(particle_positions_device);
  hipFree(particle_velocities_device);
  hipFree(particle_predicted_positions_device);
  hipFree(delta_p_device);
  hipFree(lambda_device);
  hipFree(neighbor_search_results_dev);
  hipFree(neighbor_search_results_size_prefix_sum_dev);
}

void Fluid_cuda::init() {
  const auto num_particles = particle_positions->size();
  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;

  hipMalloc(&particle_positions_device, SIZE_REAL3_N);
  hipMemcpy(particle_positions_device, particle_positions->data(), SIZE_REAL3_N, hipMemcpyHostToDevice);

  hipMalloc(&particle_velocities_device, SIZE_REAL3_N);
  hipMemset(particle_velocities_device, 0, SIZE_REAL3_N);

  hipMalloc(&particle_predicted_positions_device, SIZE_REAL3_N);
  hipMalloc(&delta_p_device, SIZE_REAL3_N);
  hipMalloc(&lambda_device, sizeof(REAL)*num_particles);

  neighbor_search_results_dev_capacity = num_particles * default_capacity;
  hipMalloc(&neighbor_search_results_dev, sizeof(int) * neighbor_search_results_dev_capacity);
  hipMalloc(&neighbor_search_results_size_prefix_sum_dev, sizeof(int) * num_particles);

  neighbor_search_results_host.resize(neighbor_search_results_dev_capacity);
  neighbor_search_results_size_prefix_sum_host.resize(num_particles);

  hipDeviceSynchronize();
  nsearch.add_point_set(
    this->particle_positions->front().data(), 
    this->particle_positions->size(), true, true
  );
  nsearch.find_neighbors();
}

void Fluid_cuda::find_neighbors(){
  int num_particles = particle_positions->size();
  nsearch.find_neighbors();

  // serial calculate prefix_sum
  for (int i = 0; i < num_particles; i++) {
    auto &pointSet = nsearch.point_set(0);
    auto count = pointSet.n_neighbors(0, i);
    const auto last_sum = i == 0 ? 0 :  neighbor_search_results_size_prefix_sum_host[i-1];
    neighbor_search_results_size_prefix_sum_host[i] = last_sum + count;
    // range for result_i: [sum_{i-1}, sum_{i})
  }

  // ensure capacity
  const auto minCapacity = neighbor_search_results_size_prefix_sum_host.back();
  if (minCapacity > neighbor_search_results_dev_capacity) {
    neighbor_search_results_dev_capacity = minCapacity * 1.2;
    neighbor_search_results_host.resize(neighbor_search_results_dev_capacity);
    hipFree(neighbor_search_results_dev);
    hipMalloc(&neighbor_search_results_dev, sizeof(int) * neighbor_search_results_dev_capacity);
  }

  for (int i = 0; i < num_particles; i++) {
    // line 6: find neighboring particles
    auto &pointSet = nsearch.point_set(0);
    auto count = pointSet.n_neighbors(0, i);
    const int start = i == 0 ? 0 :  neighbor_search_results_size_prefix_sum_host[i-1];

    memcpy(&neighbor_search_results_host[start], pointSet.neighbor_list(0, i), sizeof(int)*count);
  }

  // copy neighbor results to device
  hipMemcpy(
    neighbor_search_results_dev, 
    neighbor_search_results_host.data(), 
    sizeof(int) * neighbor_search_results_dev_capacity, 
    hipMemcpyHostToDevice
  );
  // update size prefix sum to device
  hipMemcpy(
    neighbor_search_results_size_prefix_sum_dev, 
    neighbor_search_results_size_prefix_sum_host.data(), 
    sizeof(int) * num_particles, 
    hipMemcpyHostToDevice
  );

  hipDeviceSynchronize();
}

void Fluid_cuda::simulate(REAL delta_t,
  const FluidParameters *fp,
  thrust::device_vector<Plane_cuda> &collision_objects) {
  int num_particles = particle_positions->size();
  const auto particle_positions_dev = REAL3AsVector3R(particle_positions_device);
  const auto particle_predicted_positions = REAL3AsVector3R(particle_predicted_positions_device);
  const auto particle_velocities = REAL3AsVector3R(particle_velocities_device);
  const auto delta_p = REAL3AsVector3R(delta_p_device);
  const auto density = fp->density;
  const auto particle_mass = fp->particle_mass;
  const auto damping = fp->damping;
  const auto solverIterations = fp->solverIterations;
  const auto h = fp->h;
  const auto epsilon = fp->epsilon;
  const auto n = fp->n;
  const auto k = fp->k;
  const auto c = fp->c;
  const Vector3R &external_accelerations = fp->external_forces;

  simulate_update_position_predict_position<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev, 
    particle_predicted_positions, 
    particle_velocities, 
    external_accelerations, delta_t
  );

  find_neighbors();

  for (int iter = 0; iter < solverIterations; iter++) {
    calculate_lambda<<<num_particles,1>>>(
      num_particles, 
      particle_positions_dev,
      neighbor_search_results_dev,
      neighbor_search_results_size_prefix_sum_dev,
      particle_mass, density, epsilon, h,
      lambda_device
    );

    calculate_delta_pi_and_collision_response<<<num_particles,1>>>(
      num_particles,
      particle_positions_dev,
      neighbor_search_results_dev,
      neighbor_search_results_size_prefix_sum_dev,
      delta_p,
      n, k, h, density,
      lambda_device,
      collision_objects.size(),
      thrust::raw_pointer_cast(collision_objects.data())
    );

    update_predicted_positions<<<num_particles,1>>>(
      num_particles, 
      particle_predicted_positions, 
      delta_p
    );
  }

  update_velocities<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev,
    particle_predicted_positions,
    particle_velocities,
    delta_t
  );

  apply_XSPH_viscosity<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev,
    particle_velocities,
    neighbor_search_results_dev,
    neighbor_search_results_size_prefix_sum_dev,
    particle_mass, h, c, delta_t
  );

  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;
  copy_predicted_positions_to_position(particle_positions_device, particle_predicted_positions_device, SIZE_REAL3_N);
  // copy result back to host
  hipMemcpy(particle_positions->data(), particle_positions_device, SIZE_REAL3_N, hipMemcpyDeviceToHost);
}

#endif