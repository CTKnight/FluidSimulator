#include "hip/hip_runtime.h"
#include "cudaFluid.cuh"
#include <cmath>
#include <cstdio>

#ifdef BUILD_CUDA

__global__ void simulate_update_position_predict_position(
  int n,
  Vector3R *particle_positions, 
  Vector3R *particle_preditced_position, 
  Vector3R *particle_velocities, 
  const Vector3R &external_accelerations, 
  REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    auto &positions_i = particle_positions[i];
    auto &velocities_i = particle_velocities[i];
    auto &preditced_positions_i = particle_preditced_position[i];
    if (i == 0) {
      printf("Thread 0 position: (%lf, %lf, %lf), acc: (%lf, %lf, %lf), delta_t: %lf\n", 
        positions_i.x, positions_i.y, positions_i.z,
        external_accelerations.x, external_accelerations.y, external_accelerations.z,
        delta_t
      );
    }
    velocities_i += external_accelerations * delta_t;
    preditced_positions_i = positions_i + velocities_i * delta_t;
  }
}

void copy_predicted_positions_to_position(
  REAL3 *particle_position,
  REAL3 *particle_preditced_position, 
  size_t N
) {
  hipMemcpy(particle_position, particle_preditced_position, N, hipMemcpyDeviceToDevice);
}

Fluid_cuda::Fluid_cuda(
  unique_ptr<vector<REAL3>> &&particle_positions,
  REAL h
): nsearch(h) {
  if (particle_positions == nullptr) {
    throw std::runtime_error("particle_positions == nullptr!");
  }
  this->particle_positions = std::move(particle_positions);
}

Fluid_cuda::~Fluid_cuda(){
  hipFree(particle_positions_device);
  hipFree(particle_velocities_device);
  hipFree(particle_preditced_positions_device);
  hipFree(delta_p_device);
  hipFree(lambda_device);
  const auto num_particles = particle_positions->size();

  for (int i = 0; i < num_particles; i++) {
    hipFree(neighbor_search_results_host[i]);
  }
  hipFree(neighbor_search_results_dev);
}

void Fluid_cuda::init() {
  const auto num_particles = particle_positions->size();
  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;

  hipMalloc(&particle_positions_device, SIZE_REAL3_N);
  hipMemcpy(particle_positions_device, particle_positions->data(), SIZE_REAL3_N, hipMemcpyHostToDevice);

  hipMalloc(&particle_velocities_device, SIZE_REAL3_N);
  hipMemset(particle_velocities_device, 0, SIZE_REAL3_N);

  hipMalloc(&particle_preditced_positions_device, SIZE_REAL3_N);
  hipMalloc(&delta_p_device, SIZE_REAL3_N);
  hipMalloc(&lambda_device, sizeof(REAL)*num_particles);

  hipMalloc(&neighbor_search_results_dev, sizeof(int *) * num_particles);

  neighbor_search_results_host.resize(num_particles);
  neighbor_search_results_size_host.resize(num_particles);
  std::fill(
    neighbor_search_results_size_host.begin(), 
    neighbor_search_results_size_host.end(), 
    0
  );
  neighbor_search_results_capacity_host.resize(num_particles);
  std::fill(
    neighbor_search_results_capacity_host.begin(), 
    neighbor_search_results_capacity_host.end(), 
    default_capacity
  );

  for (int i = 0; i < num_particles; i++) {
    hipMalloc(&neighbor_search_results_host[i], sizeof(int)*default_capacity);
  }
  hipMemcpy(neighbor_search_results_dev, neighbor_search_results_host.data(), sizeof(int *) * num_particles, hipMemcpyHostToDevice);
  hipMemcpy(neighbor_search_results_size_dev, neighbor_search_results_size_host.data(), sizeof(int) * num_particles, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  nsearch.add_point_set(
    this->particle_positions->front().data(), 
    this->particle_positions->size(), true, true
  );
  nsearch.find_neighbors();
}

void Fluid_cuda::find_neighbors(){
  int num_particles = particle_positions->size();
  nsearch.find_neighbors();
  for (int i = 0; i < num_particles; i++) {
    // line 6: find neighboring particles
    auto &pointSet = nsearch.point_set(0);
    auto count = pointSet.n_neighbors(0, i);
    int currentCap = neighbor_search_results_capacity_host[i];
    // if it exceeds current device array capacity
    if (count > currentCap) {
      hipFree(neighbor_search_results_host[i]);
      int newCap = static_cast<int>(count * 1.5);
      hipMalloc(&neighbor_search_results_host[i], sizeof(int)*newCap);
    }
    // update size in host
    neighbor_search_results_size_host[i] = count;
    // copy into device
    hipMemcpy(neighbor_search_results_host[i], pointSet.neighbor_list(0, i), sizeof(int)*count, hipMemcpyHostToDevice);
  }
  // update size to device
  hipMemcpy(neighbor_search_results_size_dev, neighbor_search_results_size_host.data(), sizeof(int) * num_particles, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
}

void Fluid_cuda::simulate(REAL delta_t,
  const FluidParameters *fp,
  vector<CollisionObject *> *collision_objects) {
  int num_particles = particle_positions->size();
  const auto particle_positions_dev = REAL3AsVector3R(particle_positions_device);
  const auto particle_preditced_positions = REAL3AsVector3R(particle_preditced_positions_device);
  const auto particle_velocities = REAL3AsVector3R(particle_velocities_device);
  simulate_update_position_predict_position<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev, 
    particle_preditced_positions, 
    particle_velocities, 
    fp->external_forces, delta_t
  );
  find_neighbors();
  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;
  copy_predicted_positions_to_position(particle_positions_device, particle_preditced_positions_device, SIZE_REAL3_N);
  // copy result back to host
  hipMemcpy(particle_positions->data(), particle_positions_device, SIZE_REAL3_N, hipMemcpyDeviceToHost);
}

#endif