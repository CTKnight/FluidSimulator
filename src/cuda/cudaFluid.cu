// #include <cuda>

#include "cudaFluid.cuh"

#ifdef BUILD_CUDA

__host__ __device__ void simulate_update_position() {

}

Fluid_cuda::Fluid_cuda(
  unique_ptr<vector<REAL3>> &&particle_positions,
  REAL h
): nsearch(h) {
  if (particle_positions == nullptr) {
    throw std::runtime_error("particle_positions == nullptr!");
  }
  this->particle_positions = std::move(particle_positions);
}

Fluid_cuda::~Fluid_cuda(){
  hipFree(particle_positions_device);
  hipFree(particle_velocities_device);
  hipFree(particle_preditced_position_device);
  hipFree(delta_p_device);
  hipFree(lambda_device);
  hipFree(num_particles_dev);
  const auto num_particle = particle_positions->size();

  for (int i = 0; i < num_particle; i++) {
    hipFree(neighbor_search_results_host[i]);
  }
  delete[] neighbor_search_results_host;
  hipFree(neighbor_search_results_dev);
}

void Fluid_cuda::init() {
  const auto num_particle = particle_positions->size();
  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particle; 
  hipMalloc(&particle_positions_device, SIZE_REAL3_N);

  hipMalloc(&particle_velocities_device, SIZE_REAL3_N);
  hipMemset(particle_velocities_device, 0, SIZE_REAL3_N);

  hipMalloc(&particle_preditced_position_device, SIZE_REAL3_N);
  hipMalloc(&delta_p_device, SIZE_REAL3_N);
  hipMalloc(&lambda_device, sizeof(REAL)*num_particle);

  hipMalloc(&num_particles_dev, sizeof(int));
  hipMemcpy(num_particles_dev, &num_particle, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&neighbor_search_results_dev, sizeof(int *) * num_particle);
  neighbor_search_results_host = new int*[num_particle];
  // size, capacity(include overheads of this 2 meta-elements)
  constexpr default_capacity = 50;
  int[] init_search_result = {0, default_capacity};
  for (int i = 0; i < num_particle; i++) {
    hipMalloc(&neighbor_search_results_host[i], sizeof(int)*default_capacity);
    hipMemcpy(neighbor_search_results_host[i], init_search_result, sizeof(int)*2, hipMemcpyHostToDevice);
  }
  hipMemcpy(neighbor_search_results_dev, neighbor_search_results_host, sizeof(int *) * num_particle, hipMemcpyHostToDevice);
}

void Fluid_cuda::simulate(REAL frames_per_sec, REAL simulation_steps,
  const FluidParameters *cp,
  vector<CollisionObject *> *collision_objects) {
  
}

#endif