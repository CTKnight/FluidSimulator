#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>

#include "cudaFluid.cuh"
#include "../kernel.h"

#ifdef BUILD_CUDA

__global__ void simulate_update_position_predict_position(
  int n,
  Vector3R *particle_positions, 
  Vector3R *particle_preditced_position, 
  Vector3R *particle_velocities, 
  const Vector3R external_accelerations, 
  REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    auto &positions_i = particle_positions[i];
    auto &velocities_i = particle_velocities[i];
    auto &preditced_positions_i = particle_preditced_position[i];
    if (i == 0) {
      printf("Thread 0 position: (%lf, %lf, %lf), acc: (%lf, %lf, %lf), delta_t: %lf\n", 
        positions_i.x, positions_i.y, positions_i.z,
        external_accelerations.x, external_accelerations.y, external_accelerations.z,
        delta_t
      );
    }
    velocities_i += external_accelerations * delta_t;
    preditced_positions_i = positions_i + velocities_i * delta_t;
  }
}

__global__ void calculate_lambda(
  int n,
  Vector3R *particle_positions,
  int **neighbor_search_results,
  int *neighbor_results_size,
  REAL particle_mass,
  REAL density,
  REAL epsilon,
  REAL h,
  REAL *lambda
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    // line 10: calculate lambda
    const int *neighbors_i = neighbor_search_results[i];
    const int neighbors_size_i = neighbor_results_size[i];
    // Eq 2
    REAL rho_i = 0;
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      rho_i += W_poly6(p_i-p_j, h);
    }
    // add itself
    rho_i += W_poly6(p_i-p_i, h);
    rho_i *= particle_mass;
    // Eq 1
    const REAL C_i = rho_i / density - 1.;
    REAL C_i_p_k_2_sum = 0;
    // Eq 8
    // if k = j
    REAL C_i_p_k_j_2 = 0;
    // if k = i
    Vector3R C_i_p_k_i;
    for (int jj = 0; jj < neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      const auto W_spiky_gradient_i_j = W_spiky_gradient(p_i-p_j, h) * (p_i-p_j);
      C_i_p_k_i += W_spiky_gradient_i_j;
      C_i_p_k_j_2 += W_spiky_gradient_i_j.norm2();
    }
    C_i_p_k_2_sum += C_i_p_k_i.norm2();
    C_i_p_k_2_sum /= pow(density, 2);
    lambda[i] = - C_i / (C_i_p_k_2_sum+epsilon);
  }
}

__global__ void calculate_delta_pi_and_collision_response(
  int num_particles,
  Vector3R *particle_positions,
  int **neighbor_search_results,
  int *neighbor_results_size,
  Vector3R *delta_p,
  REAL n,
  REAL k,
  REAL h,
  REAL density,
  REAL *lambda,
  CollisionObject **collision_objects
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    // line 13: calculate delta p_i
    const int *neighbors_i = neighbor_search_results[i];
    const int neighbors_size_i = neighbor_results_size[i];
    delta_p[i] = 0;
    const auto lambda_i = lambda[i];
    // Eq 12
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      // Eq 13
      double s_corr = -k*pow(W_poly6(p_i-p_j, h)/W_poly6(0.3*h, h), n);
      delta_p[i] += (lambda_i+lambda[j]+s_corr) * W_spiky_gradient(p_i-p_j, h) 
        * (p_i-p_j);
    }
    delta_p[i] /= density;
    // line 14: collision detection and response
    // TODO: apply them
    // for (const auto co: *collision_objects) {
    //   co->collide(particle_positions[i],delta_p[i]);
    // }
  }
}

__global__ void update_predicted_positions(
  int num_particles,
  Vector3R *particle_predicted_positions,
  Vector3R *delta_p
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    particle_predicted_positions[i] += delta_p[i];
  }
}

__global__ void update_velocities(
  int num_particles,
  Vector3R *particle_positions,
  Vector3R *predicted_positions,
  Vector3R *particle_velocities,
  REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    particle_velocities[i] = (predicted_positions[i] - particle_positions[i]) / delta_t;
  }
}

__global__ void apply_XSPH_viscosity(
  int num_particles,
  Vector3R *particle_positions,
  Vector3R *particle_velocities,
  int **neighbor_search_results,
  int *neighbor_results_size,
  REAL particle_mass, REAL h, REAL c, REAL delta_t
) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_particles; i += blockDim.x * gridDim.x) {
    const auto &p_i = particle_positions[i];
    const int *neighbors_i = neighbor_search_results[i];
    const int neighbors_size_i = neighbor_results_size[i];
    // line 22: vorticity confinement and XSPH viscosity
    Vector3R f_vorticity;
    Vector3R omega_i;
    // Eq 17:
    Vector3R V_xsph;
    for (int jj = 0; jj <  neighbors_size_i; jj++) {
      int j = neighbors_i[jj];
      const auto &p_j = particle_positions[j];
      const auto &p_ij = p_i-p_j;
      const auto &v_ij = particle_velocities[j] - particle_velocities[i];
      // the smallest |p_ij| with h=0.1 gives >100 so c has to correct it to ~1
      V_xsph  += v_ij * W_poly6(p_ij, h);
      omega_i += cross(v_ij, W_spiky_gradient(p_ij, h)*p_ij);
    }
    // TODO: vorticity
    // const auto &eta = ;
    // const auto &N = eta.unit();
    // f_vorticity = epsilon*cross(N, omega_i);
    V_xsph *= c;
    particle_velocities[i] += V_xsph + f_vorticity / particle_mass * delta_t;
  }
}

void copy_predicted_positions_to_position(
  REAL3 *particle_position,
  REAL3 *particle_preditced_position, 
  size_t N
) {
  hipMemcpy(particle_position, particle_preditced_position, N, hipMemcpyDeviceToDevice);
}

Fluid_cuda::Fluid_cuda(
  unique_ptr<vector<REAL3>> &&particle_positions,
  REAL h
): nsearch(h) {
  if (particle_positions == nullptr) {
    throw std::runtime_error("particle_positions == nullptr!");
  }
  this->particle_positions = std::move(particle_positions);
}

Fluid_cuda::~Fluid_cuda(){
  hipFree(particle_positions_device);
  hipFree(particle_velocities_device);
  hipFree(particle_predicted_positions_device);
  hipFree(delta_p_device);
  hipFree(lambda_device);
  const auto num_particles = particle_positions->size();

  for (int i = 0; i < num_particles; i++) {
    hipFree(neighbor_search_results_host[i]);
  }
  hipFree(neighbor_search_results_dev);
}

void Fluid_cuda::init() {
  const auto num_particles = particle_positions->size();
  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;

  hipMalloc(&particle_positions_device, SIZE_REAL3_N);
  hipMemcpy(particle_positions_device, particle_positions->data(), SIZE_REAL3_N, hipMemcpyHostToDevice);

  hipMalloc(&particle_velocities_device, SIZE_REAL3_N);
  hipMemset(particle_velocities_device, 0, SIZE_REAL3_N);

  hipMalloc(&particle_predicted_positions_device, SIZE_REAL3_N);
  hipMalloc(&delta_p_device, SIZE_REAL3_N);
  hipMalloc(&lambda_device, sizeof(REAL)*num_particles);

  hipMalloc(&neighbor_search_results_dev, sizeof(int *) * num_particles);
  hipMalloc(&neighbor_search_results_size_dev, sizeof(int) * num_particles);

  neighbor_search_results_host.resize(num_particles);
  neighbor_search_results_size_host.resize(num_particles);
  std::fill(
    neighbor_search_results_size_host.begin(), 
    neighbor_search_results_size_host.end(), 
    0
  );
  neighbor_search_results_capacity_host.resize(num_particles);
  std::fill(
    neighbor_search_results_capacity_host.begin(), 
    neighbor_search_results_capacity_host.end(), 
    default_capacity
  );

  for (int i = 0; i < num_particles; i++) {
    hipMalloc(&neighbor_search_results_host[i], sizeof(int)*default_capacity);
  }
  hipMemcpy(neighbor_search_results_dev, neighbor_search_results_host.data(), sizeof(int *) * num_particles, hipMemcpyHostToDevice);
  hipMemcpy(neighbor_search_results_size_dev, neighbor_search_results_size_host.data(), sizeof(int) * num_particles, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  nsearch.add_point_set(
    this->particle_positions->front().data(), 
    this->particle_positions->size(), true, true
  );
  nsearch.find_neighbors();
}

void Fluid_cuda::find_neighbors(){
  int num_particles = particle_positions->size();
  nsearch.find_neighbors();
  for (int i = 0; i < num_particles; i++) {
    // line 6: find neighboring particles
    auto &pointSet = nsearch.point_set(0);
    auto count = pointSet.n_neighbors(0, i);
    int currentCap = neighbor_search_results_capacity_host[i];
    // if it exceeds current device array capacity
    if (count > currentCap) {
      hipFree(neighbor_search_results_host[i]);
      int newCap = static_cast<int>(count * 1.5);
      hipMalloc(&neighbor_search_results_host[i], sizeof(int)*newCap);
    }
    // update size in host
    neighbor_search_results_size_host[i] = count;
    // copy into device
    // TODO: make it multi-stream async / batch update
    hipMemcpy(neighbor_search_results_host[i], pointSet.neighbor_list(0, i), sizeof(int)*count, hipMemcpyHostToDevice);
  }
  // update size to device
  hipMemcpy(neighbor_search_results_size_dev, neighbor_search_results_size_host.data(), sizeof(int) * num_particles, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
}

void Fluid_cuda::simulate(REAL delta_t,
  const FluidParameters *fp,
  vector<CollisionObject *> *collision_objects) {
  int num_particles = particle_positions->size();
  const auto particle_positions_dev = REAL3AsVector3R(particle_positions_device);
  const auto particle_predicted_positions = REAL3AsVector3R(particle_predicted_positions_device);
  const auto particle_velocities = REAL3AsVector3R(particle_velocities_device);
  const auto delta_p = REAL3AsVector3R(delta_p_device);
  const auto density = fp->density;
  const auto particle_mass = fp->particle_mass;
  const auto damping = fp->damping;
  const auto solverIterations = fp->solverIterations;
  const auto h = fp->h;
  const auto epsilon = fp->epsilon;
  const auto n = fp->n;
  const auto k = fp->k;
  const auto c = fp->c;
  const Vector3R &external_accelerations = fp->external_forces;

  simulate_update_position_predict_position<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev, 
    particle_predicted_positions, 
    particle_velocities, 
    external_accelerations, delta_t
  );

  find_neighbors();

  for (int iter = 0; iter < solverIterations; iter++) {
    calculate_lambda<<<num_particles,1>>>(
      num_particles, 
      particle_positions_dev,
      neighbor_search_results_dev,
      neighbor_search_results_size_dev,
      particle_mass, density, epsilon, h,
      lambda_device
    );

    calculate_delta_pi_and_collision_response<<<num_particles,1>>>(
      num_particles,
      particle_positions_dev,
      neighbor_search_results_dev,
      neighbor_search_results_size_dev,
      delta_p,
      n, k, h, density,
      lambda_device,
      nullptr
    );

    update_predicted_positions<<<num_particles,1>>>(
      num_particles, 
      particle_predicted_positions, 
      delta_p
    );
  }

  update_velocities<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev,
    particle_predicted_positions,
    particle_velocities,
    delta_t
  );

  apply_XSPH_viscosity<<<num_particles,1>>>(
    num_particles,
    particle_positions_dev,
    particle_velocities,
    neighbor_search_results_dev,
    neighbor_search_results_size_dev,
    particle_mass, h, c, delta_t
  );

  const auto SIZE_REAL3_N = sizeof(REAL3) * num_particles;
  copy_predicted_positions_to_position(particle_positions_device, particle_predicted_positions_device, SIZE_REAL3_N);
  // copy result back to host
  hipMemcpy(particle_positions->data(), particle_positions_device, SIZE_REAL3_N, hipMemcpyDeviceToHost);
}

#endif